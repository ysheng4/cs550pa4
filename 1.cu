
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


__device__ char *match(const char *s1, const char *s2){
    if(*s1==0)
  {
    if(*s2) return(char*)NULL;
    return (char*)s1;
  }
  while(*s1)
  {
    int i=0;
    while(1)
    {
      if(s2[i]==0) return (char *)s1;
      if(s2[i]!=s1[i]) break;
      i++;
    }
    s1++;
  }
  return (char*)NULL;
}

__device__ char *copy(char *dest, char *src, int n)
{
    char *tmp = dest;
        const char *s = src; 
        while (n--) *tmp++ = *s++ ; 
        return dest;
}


__global__ void grep(char *myFile, char *myregex, char *result, int line, int width){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    char *str;
    if(i < line)
    {
        str = match(&myFile[i*width], myregex);
        if(str != NULL)
            memcpy(&result[i*width], &myFile[i*width], sizeof(char)*width);
    }
}

int main(int argc, char* argv[])
{
        char *fn = argv[1];
    char *re = argv[2];
        char **file;
    char *result;
        FILE *f;
        f = fopen(fn, "r");
        file = (char **)malloc(sizeof(char*)*1024);
    result = (char *)malloc(sizeof(char)*1024*256);
    file[0] = (char *)malloc(sizeof(char)*1024*256);        
    int i,j=1;
    
    if(re==NULL||fn==NULL){
        printf("input:file name expression\n");
        return -1;
    }
   
        
    if(f == NULL)
    {
        printf("can not open file!\n");
        return -1;
    }

            
    while(j<1024)
	{
        file[j] = file[j-1] + 256;

        fgets(file[j], 256, f);
		j++;
	}

        // Memory allocation
    char *myfile, *myregex, *myresult;
    hipMalloc((void**) &myfile, sizeof(char)*1024*256);
    hipMalloc((void**) &myregex, strlen(re));
    hipMalloc((void**) &myresult, sizeof(char)*1024*256);
        // Copying memory to device
    hipMemcpy(myfile, &file[0][0], sizeof(char)*1024*256, hipMemcpyHostToDevice);
    hipMemcpy(myregex, re,  strlen(re), hipMemcpyHostToDevice);
        // Calling the kernel
    grep<<<ceil((double)1024/256), 256>>>(myfile, myregex, myresult, 1024, 256);
        // Copying results back to host
    hipMemcpy(result, myresult, sizeof(char)*1024*256, hipMemcpyDeviceToHost);
    

    for(i = 0; i < 1024; i++)
    {
        if(&result[i*256] != NULL)
            printf("%s", &result[i*256]);
    }

    return 0;
        
}
