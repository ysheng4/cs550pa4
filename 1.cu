
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


__device__ char *match(const char *s1, const char *s2){
    if(*s1==0)
  {
    if(*s2) return(char*)NULL;
    return (char*)s1;
  }
  while(*s1)
  {
    int i=0;
    while(1)
    {
      if(s2[i]==0) return (char *)s1;
      if(s2[i]!=s1[i]) break;
      i++;
    }
    s1++;
  }
  return (char*)NULL;
}

__device__ char *copy(char *dest, char *src, int n)
{
    char *tmp = dest;
        const char *s = src; 
        while (n--) *tmp++ = *s++ ; 
        return dest;
}


__global__ void grep(char *myFile, char *myregex, char *result, int line, int width){
		
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    char *ph;	 
    
    if(i < line)
    {
        ph = match(&myFile[i*width], myregex);
        if(ph != NULL)
            copy(&result[i*width], &myFile[i*width], sizeof(char)*width);
    }


}

int main(int argc, char* argv[])
{
        char *fn = argv[1];
		char *re = argv[2];
        char **file;
	    char *result;
        FILE *f;
       f = fopen(fn, "r");  
	   file = (char **)malloc(sizeof(char*)*1024);
		 file[0] = (char *)malloc(sizeof(char)*1024*256); 

        
		result = (char *)malloc(sizeof(char)*1024*256);
		       
		int i=0,j;
   
	if(re==NULL||fn==NULL){
        printf("input file or string");
        return -1;
    }
         
    while(i<1024){
        file[i] = file[i-1] + 256;
		fgets(file[i], 256, f);
		i++;
	}
        // Memory allocation
    char *myfile, *myregex, *myresult;
    hipMalloc((void**) &myfile, sizeof(char)*1024*256);
    hipMalloc((void**) &myregex, strlen(re));
    hipMalloc((void**) &myresult, sizeof(char)*1024*256);
        // Copying memory to device
    hipMemcpy(myfile, &file[0][0], sizeof(char)*1024*256, hipMemcpyHostToDevice);
    hipMemcpy(myregex, re,  strlen(re), hipMemcpyHostToDevice);
        // Calling the kernel
    grep<<<ceil((double)1024/256), 256>>>(myfile, myregex, myresult, 1024, 256);
        // Copying results back to host
    hipMemcpy(result, myresult, sizeof(char)*1024*256, hipMemcpyDeviceToHost);
    

    for(j = 0;  j< 1024; j++)
    {
        if(&result[j*256] != NULL)
            printf("%s", &result[j*256]);
    }

    return 0;
        
}
