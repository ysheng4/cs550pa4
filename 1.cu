
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


__device__ char *matchstring(const char *s1, const char *s2){
    if(*s1==0)
  {
    if(*s2) return(char*)NULL;
    return (char*)s1;
  }
  while(*s1)
  {
    int i=0;
    while(1)
    {
      if(s2[i]==0) return (char *)s1;
      if(s2[i]!=s1[i]) break;
      i++;
    }
    s1++;
  }
  return (char*)NULL;
}

__device__ char *matchcopy(char *dest, char *src, int n)
{
    char *tmp = dest;
        const char *s = src; 
        while (n--) *tmp++ = *s++ ; 
        return dest;
}


__global__ void grep(char *myFile, char *myregex, char *result, int line, int width){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    char *ph;
    if(i < line)
    {
        ph = matchstring(&myFile[i*width], myregex);
        if(ph != NULL)
            matchcopy(&result[i*width], &myFile[i*width], sizeof(char)*width);
    }
}

int main(int argc, char* argv[])
{
        char *fn = argv[1];
    char *re = argv[2];
        char **file;
    char *result;
        FILE *f;
        f = fopen(fn, "r");
        file = (char **)malloc(sizeof(char*)*1024);
    result = (char *)malloc(sizeof(char)*1024*256);
    file[0] = (char *)malloc(sizeof(char)*1024*256);        
    int i;
    
    if(re==NULL||fn==NULL){
        printf("input:file name expression\n");
        return -1;
    }
   
        
    if(f == NULL)
    {
        printf("can not open file!\n");
        return -1;
    }

            
    for(i = 1; i < 1024; i++)
	{
        file[i] = file[i-1] + 256;

        fgets(file[i], 256, f);
	}

        // Memory allocation
    char *myfile, *myregex, *myresult;
    hipMalloc((void**) &myfile, sizeof(char)*1024*256);
    hipMalloc((void**) &myregex, strlen(re));
    hipMalloc((void**) &myresult, sizeof(char)*1024*256);
        // Copying memory to device
    hipMemcpy(myfile, &file[0][0], sizeof(char)*1024*256, hipMemcpyHostToDevice);
    hipMemcpy(myregex, re,  strlen(re), hipMemcpyHostToDevice);
        // Calling the kernel
    grep<<<ceil((double)1024/256), 256>>>(myfile, myregex, myresult, 1024, 256);
        // Copying results back to host
    hipMemcpy(result, myresult, sizeof(char)*1024*256, hipMemcpyDeviceToHost);
    

    for(i = 0; i < 1024; i++)
    {
        if(&result[i*256] != NULL)
            printf("%s", &result[i*256]);
    }

    return 0;
        
}
