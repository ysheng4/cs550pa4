
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


__device__ char *match(const char *s1, const char *s2){
    if(*s1==0)
  {
    if(*s2) return(char*)NULL;
    return (char*)s1;
  }
  while(*s1)
  {
    int i=0;
    while(1)
    {
      if(s2[i]==0) return (char *)s1;
      if(s2[i]!=s1[i]) break;
      i++;
    }
    s1++;
  }
  return (char*)NULL;
}


__global__ void grep(char *myfile, char *myregex, char *result, int line, int width){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    char *str;
    if(i < line)
    {
        str = match(&myfile[i*width], myregex);
        if(str != NULL)
            memcpy(&result[i*width], &myfile[i*width], sizeof(char)*width);
    }
}

int main(int argc, char* argv[])
{
	int i,j=1;
    char *fn = argv[1],*re = argv[2];
    char **file = (char **)malloc(sizeof(char*)*1024);
    char *result= (char *)malloc(sizeof(char)*1024*256);
	char *myfile, *myregex, *myresult;
    FILE *f;
    f = fopen(fn, "r");
    file[0] = (char *)malloc(sizeof(char)*1024*256);        
    
    

    if(re==NULL||fn==NULL){
        printf("input string or file");
        return -1;
    }
   
            
    while(j<1024)
	{
        file[j] = file[j-1] + 256;

        fgets(file[j], 256, f);
		j++;
	}

        // Memory allocation
    
    hipMalloc((void**) &myfile, sizeof(char)*1024*256);
    hipMalloc((void**) &myregex, strlen(re));
    hipMalloc((void**) &myresult, sizeof(char)*1024*256);
        // Copying memory to device
    hipMemcpy(myfile, &file[0][0], sizeof(char)*1024*256, hipMemcpyHostToDevice);
    hipMemcpy(myregex, re,  strlen(re), hipMemcpyHostToDevice);
        // Calling the kernel
    grep<<<ceil((double)1024/256), 256>>>(myfile, myregex, myresult, 1024, 256);
        // Copying results back to host
    hipMemcpy(result, myresult, sizeof(char)*1024*256, hipMemcpyDeviceToHost);
  	hipFree(myfile);    
	hipFree(myregex);
	hipFree(myresult);  

    for(i = 0; i < 1024; i++)
    {
        if(&result[i*256] != NULL)
            printf("%s", &result[i*256]);
    }


    return 0;
	
        
}
