
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


__device__ char *match(const char *s1, const char *s2){
    if(*s1==0)
  {
    if(*s2) return(char*)NULL;
    return (char*)s1;
  }
  while(*s1)
  {
    int i=0;
    while(1)
    {
      if(s2[i]==0) return (char *)s1;
      if(s2[i]!=s1[i]) break;
      i++;
    }
    s1++;
  }
  return (char*)NULL;
}


__global__ void grep(char *myfile, char *mystring, char *result, int line, int width){
    int j=0,count=0;
    char *str;
	 while(j<1024)
		   {
			 str = match(&myfile[j*256], mystring);
			 if(str != NULL)
			 {
				memcpy(&result[count*256], &myfile[j*256], sizeof(char)*256);
				count++;			
			 }
				j++;
		   }

}

int main(int argc, char* argv[])
{
	int i=1,j=0;
    char *fn = argv[1],*re = argv[2];
    char **file = (char **)malloc(sizeof(char*)*1024);
    char *result= (char *)malloc(sizeof(char)*1024*256);
	char *myfile, *mystring, *myresult;
    FILE *f;
    f = fopen(fn, "r");
    file[0] = (char *)malloc(sizeof(char)*1024*256);           
    if(re==NULL||fn==NULL)
	{
        printf("input string or file");
        return -1;
    }               
    while(i<1024)
	{
        file[i] = file[i-1] + 256;
        fgets(file[i], 256, f);
		i++;
	}
// Memory allocation   
    hipMalloc((void**) &myfile, sizeof(char)*1024*256);
    hipMalloc((void**) &mystring, strlen(re));
    hipMalloc((void**) &myresult, sizeof(char)*1024*256);
// Copying memory to device
    hipMemcpy(myfile, &file[0][0], sizeof(char)*1024*256, hipMemcpyHostToDevice);
    hipMemcpy(mystring, re,  strlen(re), hipMemcpyHostToDevice);
// Calling the kernel
    grep<<<ceil((double)1024/256), 256>>>(myfile, mystring, myresult, 1024, 256);
// Copying results back to host
    hipMemcpy(result, myresult, sizeof(char)*1024*256, hipMemcpyDeviceToHost);
  	hipFree(myfile);    
	hipFree(mystring);
	hipFree(myresult);  
    for(j = 0; j < 1024; j++)
    {
        if(&result[j*256] != NULL)
            printf("%s", &result[j*256]);
    }
    return 0;        
}
